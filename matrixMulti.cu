#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
        else { \
            printf("CUDA call at %s:%d succeeded.\n", __FILE__, __LINE__); \
        } \
    } while (0)
#include <stdio.h>
#include <hip/hip_runtime.h>

// 普通矩阵乘法：C = A (MxK) * B (KxN) => C (MxN)
__global__ void matMulKernel(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int M = 512; // A的行数
    int K = 256; // A的列数，B的行数
    int N = 384; // B的列数

    int sizeA = M * K * sizeof(float);
    int sizeB = K * N * sizeof(float);
    int sizeC = M * N * sizeof(float);

    float *h_A = (float*)malloc(sizeA);
    float *h_B = (float*)malloc(sizeB);
    float *h_C = (float*)malloc(sizeC);

    // 初始化矩阵
    for (int i = 0; i < M * K; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < K * N; ++i) h_B[i] = 2.0f;

    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, sizeA));
    CUDA_CHECK(hipMalloc((void**)&d_B, sizeB));
    CUDA_CHECK(hipMalloc((void**)&d_C, sizeC));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));

    // 打印部分结果
    printf("C[0]=%f\n", h_C[0]);
    printf("C[M*N-1]=%f\n", h_C[M*N-1]);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
